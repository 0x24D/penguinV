#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../../../src/image_buffer.h"
#include "../../../src/image_exception.h"
#include "../../../src/image_function.h"
#include "../../../src/cuda/cuda_types.cuh"
#include "../../../src/cuda/cuda_helper.cuh"
#include "../../../src/cuda/image_function_cuda.cuh"
#include "../unit_test_helper.h"
#include "unit_test_helper_cuda.cuh"

namespace
{
    // This function must run with thread count as 1
    __global__ void isEqualCuda( const uint8_t * image, uint8_t value, uint32_t width, uint32_t height, uint32_t * differenceCount )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if( x < width && y < height )
        {
            const uint32_t id = y * width + x;

            if( image[id] == value )
                atomicAdd( differenceCount, 1 );
        }
    }

    __global__ void isAnyEqualCuda( const uint8_t * image, uint8_t * value, size_t valueCount, uint32_t width, uint32_t height,
                                    uint32_t * differenceCount )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if( x < width && y < height )
        {
            const uint32_t id = y * width + x;

            bool equal = false;

            for( uint32_t i = 0; i < valueCount; ++i )
            {
                if( image[id] == value[i] )
                {
                    equal = true;
                    break;
                }
            }

            if( equal )
                atomicAdd( differenceCount, 1 );
        }
    }

    PenguinV_Image::Image generateImage( uint32_t width, uint32_t height, uint8_t colorCount, uint8_t value )
    {
        PenguinV_Image::ImageCuda image( width, height, colorCount );

        image.fill( value );

        PenguinV_Image::Image imageOut;
        imageOut.swap( image );

        return imageOut;
    }
}

namespace Unit_Test
{
    namespace Cuda
    {
        PenguinV_Image::Image uniformImage( uint8_t value )
        {
            return generateImage( randomValue<uint32_t>( 1, 2048 ), randomValue<uint32_t>( 1, 2048 ), PenguinV_Image::GRAY_SCALE, value );
        }

        PenguinV_Image::Image uniformImage()
        {
            return uniformImage( randomValue<uint8_t>( 256 ) );
        }

        PenguinV_Image::Image uniformRGBImage()
        {
            return uniformRGBImage( randomValue<uint8_t>( 256 ) );
        }

        PenguinV_Image::Image uniformRGBImage( uint8_t value )
        {
            return generateImage( randomValue<uint32_t>( 1, 2048 ), randomValue<uint32_t>( 1, 2048 ), PenguinV_Image::RGB, value );
        }

        PenguinV_Image::Image blackImage()
        {
            return uniformImage( 0u );
        }

        PenguinV_Image::Image whiteImage()
        {
            return uniformImage( 255u );
        }

        std::vector < PenguinV_Image::Image > uniformImages( uint32_t images )
        {
            if( images == 0 )
                throw imageException( "Invalid parameter" );

            std::vector < PenguinV_Image::Image > image;

            image.push_back( uniformImage() );

            image.resize( images );

            for( size_t i = 1; i < image.size(); ++i ) {
                image[i] = image.front().generate( image[0].width(), image[0].height() );
                image[i].fill( randomValue<uint8_t>( 256 ) );
            }

            return image;
        }

        std::vector < PenguinV_Image::Image > uniformImages( std::vector < uint8_t > intensityValue )
        {
            if( intensityValue.size() == 0 )
                throw imageException( "Invalid parameter" );

            std::vector < PenguinV_Image::Image > image;

            image.push_back( uniformImage( intensityValue[0] ) );

            image.resize( intensityValue.size() );

            for( size_t i = 1; i < image.size(); ++i ) {
                image[i] = image.front().generate( image[0].width(), image[0].height() );
                image[i].fill( intensityValue[i] );
            }

            return image;
        }

        bool verifyImage( const PenguinV_Image::Image & image, uint8_t value )
        {
            multiCuda::Type<uint32_t> differenceCount( 0 );

            const uint32_t rowSize = image.rowSize();
            const uint32_t height = image.height();

            launchKernel2D( isEqualCuda, rowSize, height,
                            image.data(), value, rowSize, height, differenceCount.data() );

            return differenceCount.get() == rowSize * height;
        }

        bool verifyImage( const PenguinV_Image::Image & image, const std::vector < uint8_t > & value )
        {
            multiCuda::Type<uint32_t> differenceCount( 0 );
            multiCuda::Array<uint8_t> valueCuda( value );

            const uint32_t rowSize = image.rowSize();
            const uint32_t height = image.height();

            launchKernel2D( isAnyEqualCuda, rowSize, height,
                            image.data(), valueCuda.data(), valueCuda.size(), rowSize, height, differenceCount.data() );

            return differenceCount.get() == rowSize * height;
        }
    }
}
