#include <hip/hip_runtime.h>
#include "cuda_helper.cuh"
#include "../image_exception.h"

namespace Cuda
{
    bool isCudaSupported()
    {
        int deviceCount = 0;
        if( !cudaSafeCheck( hipGetDeviceCount( &deviceCount ) ) )
            return false;

        return (deviceCount > 0);
    }

    void validateKernel()
    {
        hipError_t error = hipGetLastError();
        if( error != hipSuccess )
            throw imageException( "Failed to launch CUDA kernel" );
    }

    void cudaCheck( hipError_t error )
    {
        if( error != hipSuccess ) {
            hipGetLastError();

            char errorMessage[64];
            sprintf( errorMessage, "Failed to run CUDA function with error %d", error );

            throw imageException( errorMessage );
        }
    }

    bool cudaSafeCheck( hipError_t error )
    {
        const bool sucess = (error == hipSuccess);
        if( !sucess )
            hipGetLastError();

        return sucess;
    }

    KernelParameters::KernelParameters( uint32_t threadsPerBlock_, uint32_t blocksPerGrid_ )
        : threadsPerBlock( threadsPerBlock_ )
        , blocksPerGrid  ( blocksPerGrid_ )
    {
    }

    KernelParameters getKernelParameters( uint32_t size )
    {
        static const uint32_t threadsPerBlock = 256;
        return KernelParameters( threadsPerBlock, (size + threadsPerBlock - 1) / threadsPerBlock );
    }
}
