#include <hip/hip_runtime.h>
#include "cuda_helper.cuh"
#include "../image_exception.h"

KernelParameters::KernelParameters( uint32_t threadsPerBlock_, uint32_t blocksPerGrid_ )
    : threadsPerBlock( threadsPerBlock_ )
    , blocksPerGrid  ( blocksPerGrid_ )
{
}

KernelParameters getKernelParameters( uint32_t size )
{
    static const uint32_t threadsPerBlock = 256;
    return KernelParameters( threadsPerBlock, (size + threadsPerBlock - 1) / threadsPerBlock );
}

void ValidateLastError()
{
    hipError_t error = hipGetLastError();
    if( error != hipSuccess )
        throw imageException( "Failed to launch CUDA kernel" );
}

void cudaCheck( hipError_t error )
{
    if( error != hipSuccess ) {
        hipGetLastError();

        char errorMessage[64];
        sprintf( errorMessage, "Failed to run CUDA function with error %d", error );

        throw imageException( errorMessage );
    }
}

bool cudaSafeCheck( hipError_t error )
{
    const bool sucess = (error == hipSuccess);
    if( !sucess )
        hipGetLastError();

    return sucess;
}

bool IsCudaSupported()
{
    int deviceCount = 0;
    if( !cudaSafeCheck( hipGetDeviceCount( &deviceCount ) ) )
        return false;

    return (deviceCount > 0);
}
