#include <hip/hip_runtime.h>
#include "cuda_helper.cuh"
#include "../image_exception.h"

KernelParameters::KernelParameters( uint32_t threadsPerBlock_, uint32_t blocksPerGrid_  )
    : threadsPerBlock( threadsPerBlock_ )
    , blocksPerGrid  ( blocksPerGrid_   )
{
}

KernelParameters getKernelParameters( uint32_t size )
{
    static const uint32_t threadsPerBlock = 256;
    return KernelParameters( threadsPerBlock, (size + threadsPerBlock - 1) / threadsPerBlock );
}

void ValidateLastError()
{
    hipError_t error = hipGetLastError();
    if( error != hipSuccess )
        throw imageException( "Failed to launch CUDA kernel" );
}
