#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include "../image_function.h"
#include "image_function_cuda.cuh"

namespace
{
    // Helper function which should return proper arguments for CUDA device functions
    void getKernelParameters( int & threadsPerBlock, int & blocksPerGrid, uint32_t size )
    {
        if( size < 256 ) {
            threadsPerBlock = size;
            blocksPerGrid = 1;
        }
        else {
            threadsPerBlock = 256;
            blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
        }
    };

    // The list of CUDA device functions
    __global__ void absoluteDifference( const uint8_t * in1, const uint8_t * in2, uint8_t * out, uint32_t size )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size ) {
            out[id] = in1[id] > in2[id] ? in1[id] - in2[id] : in2[id] - in1[id];
        }
    };

    __global__ void bitwiseAnd( const uint8_t * in1, const uint8_t * in2, uint8_t * out, uint32_t size )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size ) {
            out[id] = in1[id] & in2[id];
        }
    };

    __global__ void bitwiseOr( const uint8_t * in1, const uint8_t * in2, uint8_t * out, uint32_t size )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size ) {
            out[id] = in1[id] | in2[id];
        }
    };

    __global__ void bitwiseXor( const uint8_t * in1, const uint8_t * in2, uint8_t * out, uint32_t size )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size ) {
            out[id] = in1[id] ^ in2[id];
        }
    };

    __global__ void fill( uint8_t * data, uint8_t value, uint32_t size )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size ) {
            data[id] = value;
        }
    };

    __global__ void gammaCorrection( const uint8_t * in, uint8_t * out, uint32_t size, double a, float gamma )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        __shared__ uint8_t value[256];

        if( threadIdx.x == 0 ) {
            for( uint16_t i = 0; i < 256; ++i ) {
                double data = a * __powf( __fdividef( (float)i, 255.0f ), gamma ) * 255 + 0.5;

                if( data < 255 )
                    value[i] = static_cast<uint8_t>(data);
                else
                    value[i] = 255;
            }
        }

        __syncthreads();

        if( id < size ) {
            out[id] = value[in[id]];
        }
    };

    __global__ void invert( const uint8_t * in, uint8_t * out, uint32_t size )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size ) {
            out[id] = ~in[id];
        }
    };

    __global__ void maximum( const uint8_t * in1, const uint8_t * in2, uint8_t * out, uint32_t size )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size ) {
            out[id] = in1[id] > in2[id] ? in1[id] : in2[id];
        }
    };

    __global__ void minimum( const uint8_t * in1, const uint8_t * in2, uint8_t * out, uint32_t size )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size ) {
            out[id] = in1[id] < in2[id] ? in1[id] : in2[id];
        }
    };

    __global__ void subtract( const uint8_t * in1, const uint8_t * in2, uint8_t * out, uint32_t size )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size ) {
            out[id] = in1[id] > in2[id] ? in1[id] - in2[id] : 0;
        }
    };
};

namespace Image_Function_Cuda
{
    Image AbsoluteDifference( const Image & in1, const Image & in2 )
    {
        Image_Function::ParameterValidation( in1, in2 );

        Image out( in1.width(), in1.height() );

        AbsoluteDifference( in1, in2, out );

        return out;
    }

    void AbsoluteDifference( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function::ParameterValidation( in1, in2, out );

        int threadsPerBlock = 1, blocksPerGrid = 1;
        getKernelParameters( threadsPerBlock, blocksPerGrid, out.width() * out.height() );

        absoluteDifference<<<blocksPerGrid, threadsPerBlock>>>(in1.data(), in2.data(), out.data(), out.width() * out.height());
        hipError_t error = hipGetLastError();
        if( error != hipSuccess )
            throw imageException( "Failed to launch CUDA kernel" );
    }

    Image BitwiseAnd( const Image & in1, const Image & in2 )
    {
        Image_Function::ParameterValidation( in1, in2 );

        Image out( in1.width(), in1.height() );

        BitwiseAnd( in1, in2, out );

        return out;
    }

    void BitwiseAnd( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function::ParameterValidation( in1, in2, out );

        int threadsPerBlock = 1, blocksPerGrid = 1;
        getKernelParameters( threadsPerBlock, blocksPerGrid, out.width() * out.height() );

        bitwiseAnd<<<blocksPerGrid, threadsPerBlock>>>(in1.data(), in2.data(), out.data(), out.width() * out.height());
        hipError_t error = hipGetLastError();
        if( error != hipSuccess )
            throw imageException( "Failed to launch CUDA kernel" );
    }

    Image BitwiseOr( const Image & in1, const Image & in2 )
    {
        Image_Function::ParameterValidation( in1, in2 );

        Image out( in1.width(), in1.height() );

        BitwiseOr( in1, in2, out );

        return out;
    }

    void BitwiseOr( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function::ParameterValidation( in1, in2, out );

        int threadsPerBlock = 1, blocksPerGrid = 1;
        getKernelParameters( threadsPerBlock, blocksPerGrid, out.width() * out.height() );

        bitwiseOr<<<blocksPerGrid, threadsPerBlock>>>(in1.data(), in2.data(), out.data(), out.width() * out.height());
        hipError_t error = hipGetLastError();
        if( error != hipSuccess )
            throw imageException( "Failed to launch CUDA kernel" );
    }

    Image BitwiseXor( const Image & in1, const Image & in2 )
    {
        Image_Function::ParameterValidation( in1, in2 );

        Image out( in1.width(), in1.height() );

        BitwiseXor( in1, in2, out );

        return out;
    }

    void BitwiseXor( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function::ParameterValidation( in1, in2, out );

        int threadsPerBlock = 1, blocksPerGrid = 1;
        getKernelParameters( threadsPerBlock, blocksPerGrid, out.width() * out.height() );

        bitwiseXor<<<blocksPerGrid, threadsPerBlock>>>(in1.data(), in2.data(), out.data(), out.width() * out.height());
        hipError_t error = hipGetLastError();
        if( error != hipSuccess )
            throw imageException( "Failed to launch CUDA kernel" );
    }

    void Convert( const Bitmap_Image::Image & in, Image & out )
    {
        Image_Function::ParameterValidation( in );
        Image_Function::ParameterValidation( out );

        if( in.width() != out.width() || in.height() != out.height() )
            throw imageException( "Bad input parameters in image function" );

        uint32_t rowSizeIn  = in.rowSize();
        uint32_t rowSizeOut = out.width();

        const uint8_t * Y    = in.data();
        const uint8_t * YEnd = Y + in.height() * rowSizeIn;

        uint8_t * cudaY = out.data();

        for( ; Y != YEnd; Y += rowSizeIn, cudaY += rowSizeOut ) {
            hipError_t error = hipMemcpy( cudaY, Y, out.width() * sizeof( uint8_t ), hipMemcpyHostToDevice );
            if( error != hipSuccess )
                throw imageException( "Cannot copy a memory to CUDA device" );
        }
    }

    void Convert( const Image & in, Bitmap_Image::Image & out )
    {
        Image_Function::ParameterValidation( in );
        Image_Function::ParameterValidation( out );

        if( in.width() != out.width() || in.height() != out.height() )
            throw imageException( "Bad input parameters in image function" );

        uint32_t rowSizeIn  = in.width();
        uint32_t rowSizeOut = out.rowSize();

        uint8_t * Y    = out.data();
        const uint8_t * YEnd = Y + out.height() * rowSizeOut;

        const uint8_t * cudaY = in.data();

        for( ; Y != YEnd; Y += rowSizeOut, cudaY += rowSizeIn ) {
            hipError_t error = hipMemcpy( Y, cudaY, in.width() * sizeof( uint8_t ), hipMemcpyDeviceToHost );
            if( error != hipSuccess )
                throw imageException( "Cannot copy a memory from CUDA device" );
        }
    }

    void Copy( const Image & in, Image & out )
    {
        Image_Function::ParameterValidation( in, out );

        out = in;
    }

    void Fill( Image & image, uint8_t value )
    {
        Image_Function::ParameterValidation( image );

        int threadsPerBlock = 1, blocksPerGrid = 1;
        getKernelParameters( threadsPerBlock, blocksPerGrid, image.width() * image.height() );

        fill<<<blocksPerGrid, threadsPerBlock>>>(image.data(), value, image.width() * image.height());
        hipError_t error = hipGetLastError();
        if( error != hipSuccess )
            throw imageException( "Failed to launch CUDA kernel" );
    }

    Image GammaCorrection( const Image & in, double a, double gamma )
    {
        Image_Function::ParameterValidation( in );

        Image out( in.width(), in.height() );

        GammaCorrection( in, out, a, gamma );

        return out;
    }

    void GammaCorrection( const Image & in, Image & out, double a, double gamma )
    {
        Image_Function::ParameterValidation( in, out );

        if( a < 0 || gamma < 0 )
            throw imageException( "Bad input parameters in image function" );

        int threadsPerBlock = 1, blocksPerGrid = 1;
        getKernelParameters( threadsPerBlock, blocksPerGrid, out.width() * out.height() );

        gammaCorrection<<<blocksPerGrid, threadsPerBlock>>>(in.data(), out.data(), out.width() * out.height(), a, static_cast<float>(gamma));
        hipError_t error = hipGetLastError();
        if( error != hipSuccess )
            throw imageException( "Failed to launch CUDA kernel" );
    }

    Image Invert( const Image & in )
    {
        Image_Function::ParameterValidation( in );

        Image out( in.width(), in.height() );

        Invert( in, out );

        return out;
    }

    void Invert( const Image & in, Image & out )
    {
        Image_Function::ParameterValidation( in, out );

        int threadsPerBlock = 1, blocksPerGrid = 1;
        getKernelParameters( threadsPerBlock, blocksPerGrid, out.width() * out.height() );

        invert<<<blocksPerGrid, threadsPerBlock>>>(in.data(), out.data(), out.width() * out.height());
        hipError_t error = hipGetLastError();
        if( error != hipSuccess )
            throw imageException( "Failed to launch CUDA kernel" );
    }

    Image Maximum( const Image & in1, const Image & in2 )
    {
        Image_Function::ParameterValidation( in1, in2 );

        Image out( in1.width(), in1.height() );

        Maximum( in1, in2, out );

        return out;
    }

    void Maximum( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function::ParameterValidation( in1, in2, out );

        int threadsPerBlock = 1, blocksPerGrid = 1;
        getKernelParameters( threadsPerBlock, blocksPerGrid, out.width() * out.height() );

        maximum<<<blocksPerGrid, threadsPerBlock>>>(in1.data(), in2.data(), out.data(), out.width() * out.height());
        hipError_t error = hipGetLastError();
        if( error != hipSuccess )
            throw imageException( "Failed to launch CUDA kernel" );
    }

    Image Minimum( const Image & in1, const Image & in2 )
    {
        Image_Function::ParameterValidation( in1, in2 );

        Image out( in1.width(), in1.height() );

        Minimum( in1, in2, out );

        return out;
    }

    void Minimum( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function::ParameterValidation( in1, in2, out );

        int threadsPerBlock = 1, blocksPerGrid = 1;
        getKernelParameters( threadsPerBlock, blocksPerGrid, out.width() * out.height() );

        minimum<<<blocksPerGrid, threadsPerBlock>>>(in1.data(), in2.data(), out.data(), out.width() * out.height());
        hipError_t error = hipGetLastError();
        if( error != hipSuccess )
            throw imageException( "Failed to launch CUDA kernel" );
    }

    Image Subtract( const Image & in1, const Image & in2 )
    {
        Image_Function::ParameterValidation( in1, in2 );

        Image out( in1.width(), in1.height() );

        Subtract( in1, in2, out );

        return out;
    }

    void Subtract( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function::ParameterValidation( in1, in2, out );

        int threadsPerBlock = 1, blocksPerGrid = 1;
        getKernelParameters( threadsPerBlock, blocksPerGrid, out.width() * out.height() );

        subtract<<<blocksPerGrid, threadsPerBlock>>>(in1.data(), in2.data(), out.data(), out.width() * out.height());
        hipError_t error = hipGetLastError();
        if( error != hipSuccess )
            throw imageException( "Failed to launch CUDA kernel" );
    }
};
