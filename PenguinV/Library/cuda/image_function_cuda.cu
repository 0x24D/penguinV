#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include "cuda_types.cuh"
#include "image_function_cuda.cuh"

namespace
{
    struct KernelParameters
    {
        KernelParameters( int threadsPerBlock_, int blocksPerGrid_  )
            : threadsPerBlock( threadsPerBlock_ )
            , blocksPerGrid  ( blocksPerGrid_   )
        {};

        int threadsPerBlock;
        int blocksPerGrid;
    };

    // Helper function which should return proper arguments for CUDA device functions
    KernelParameters getKernelParameters( uint32_t size )
    {
        static const int threadsPerBlock = 256;
        return KernelParameters( threadsPerBlock, (size + threadsPerBlock - 1) / threadsPerBlock );
    };

    // Validation of last occured error in functions on host side
    void ValidateLastError()
    {
        hipError_t error = hipGetLastError();
        if( error != hipSuccess )
            throw imageException( "Failed to launch CUDA kernel" );
    };

    // The list of CUDA device functions on device side
    __global__ void absoluteDifferenceCuda( const uint8_t * in1, const uint8_t * in2, uint8_t * out, uint32_t size )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size ) {
            out[id] = in1[id] > in2[id] ? in1[id] - in2[id] : in2[id] - in1[id];
        }
    };

    __global__ void bitwiseAndCuda( const uint8_t * in1, const uint8_t * in2, uint8_t * out, uint32_t size )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size ) {
            out[id] = in1[id] & in2[id];
        }
    };

    __global__ void bitwiseOrCuda( const uint8_t * in1, const uint8_t * in2, uint8_t * out, uint32_t size )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size ) {
            out[id] = in1[id] | in2[id];
        }
    };

    __global__ void bitwiseXorCuda( const uint8_t * in1, const uint8_t * in2, uint8_t * out, uint32_t size )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size ) {
            out[id] = in1[id] ^ in2[id];
        }
    };

    __global__ void convertToGrayScaleCuda( const uint8_t * in, uint8_t * out, uint32_t size, uint32_t width, uint8_t colorCount )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size ) {
            uint32_t x = id % width;
            uint32_t y = id / width;

            uint32_t sum = 0;

            const uint8_t * data = in + (width * y + x) * colorCount;

            for( uint8_t i = 0; i < colorCount; ++i, ++data )
            {
                sum += (*data);
            }

            out[id] = static_cast<uint8_t>(static_cast<float>(sum) / colorCount + 0.5);
        }
    };

    __global__ void fillCuda( uint8_t * data, uint8_t value, uint32_t size )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size ) {
            data[id] = value;
        }
    };

    __global__ void lookupTableCuda( const uint8_t * in, uint8_t * out, uint32_t size, uint8_t * table )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size ) {
            out[id] = table[in[id]];
        }
    };

    __global__ void invertCuda( const uint8_t * in, uint8_t * out, uint32_t size )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size ) {
            out[id] = ~in[id];
        }
    };

    __global__ void maximumCuda( const uint8_t * in1, const uint8_t * in2, uint8_t * out, uint32_t size )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size ) {
            out[id] = in1[id] > in2[id] ? in1[id] : in2[id];
        }
    };

    __global__ void minimumCuda( const uint8_t * in1, const uint8_t * in2, uint8_t * out, uint32_t size )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size ) {
            out[id] = in1[id] < in2[id] ? in1[id] : in2[id];
        }
    };

    __global__ void subtractCuda( const uint8_t * in1, const uint8_t * in2, uint8_t * out, uint32_t size )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size ) {
            out[id] = in1[id] > in2[id] ? in1[id] - in2[id] : 0;
        }
    };

    __global__ void thresholdCuda( const uint8_t * in, uint8_t * out, uint32_t size, uint8_t threshold )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size ) {
            out[id] = in[id] < threshold ? 0 : 255;
        }
    };

    __global__ void thresholdCuda( const uint8_t * in, uint8_t * out, uint32_t size, uint8_t minThreshold, uint8_t maxThreshold )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size ) {
            out[id] = in[id] < minThreshold || in[id] > maxThreshold ? 0 : 255;
        }
    };
};

namespace Image_Function_Cuda
{
    Image AbsoluteDifference( const Image & in1, const Image & in2 )
    {
        Image_Function::ParameterValidation( in1, in2 );

        Image out( in1.width(), in1.height() );

        AbsoluteDifference( in1, in2, out );

        return out;
    }

    void AbsoluteDifference( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function::ParameterValidation( in1, in2, out );

        const uint32_t size = out.rowSize() * out.height();
        const KernelParameters kernel = getKernelParameters( size );

        absoluteDifferenceCuda<<<kernel.blocksPerGrid, kernel.threadsPerBlock>>>( in1.data(), in2.data(), out.data(), size );
        
        ValidateLastError();
    }

    Image BitwiseAnd( const Image & in1, const Image & in2 )
    {
        Image_Function::ParameterValidation( in1, in2 );

        Image out( in1.width(), in1.height() );

        BitwiseAnd( in1, in2, out );

        return out;
    }

    void BitwiseAnd( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function::ParameterValidation( in1, in2, out );

        const uint32_t size = out.rowSize() * out.height();
        const KernelParameters kernel = getKernelParameters( size );

        bitwiseAndCuda<<<kernel.blocksPerGrid, kernel.threadsPerBlock>>>(in1.data(), in2.data(), out.data(), size);
        
        ValidateLastError();
    }

    Image BitwiseOr( const Image & in1, const Image & in2 )
    {
        Image_Function::ParameterValidation( in1, in2 );

        Image out( in1.width(), in1.height() );

        BitwiseOr( in1, in2, out );

        return out;
    }

    void BitwiseOr( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function::ParameterValidation( in1, in2, out );

        const uint32_t size = out.rowSize() * out.height();
        const KernelParameters kernel = getKernelParameters( size );

        bitwiseOrCuda<<<kernel.blocksPerGrid, kernel.threadsPerBlock>>>(in1.data(), in2.data(), out.data(), size);
        
        ValidateLastError();
    }

    Image BitwiseXor( const Image & in1, const Image & in2 )
    {
        Image_Function::ParameterValidation( in1, in2 );

        Image out( in1.width(), in1.height() );

        BitwiseXor( in1, in2, out );

        return out;
    }

    void BitwiseXor( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function::ParameterValidation( in1, in2, out );

        const uint32_t size = out.rowSize() * out.height();
        const KernelParameters kernel = getKernelParameters( size );

        bitwiseXorCuda<<<kernel.blocksPerGrid, kernel.threadsPerBlock>>>(in1.data(), in2.data(), out.data(), size);
        
        ValidateLastError();
    }

    void Convert( const Bitmap_Image::Image & in, Image & out )
    {
        Image_Function::ParameterValidation( in );
        Image_Function::ParameterValidation( out );

        if( in.width() != out.width() || in.height() != out.height() || in.alignment() != 1u ||
            in.colorCount() != out.colorCount())
            throw imageException( "Bad input parameters in image function" );

        const uint32_t size = in.rowSize() * in.height();

        hipError_t error = hipMemcpy( out.data(), in.data(), size, hipMemcpyHostToDevice );
        if( error != hipSuccess )
            throw imageException( "Cannot copy a memory to CUDA device" );
    }

    void Convert( const Image & in, Bitmap_Image::Image & out )
    {
        Image_Function::ParameterValidation( in );
        Image_Function::ParameterValidation( out );

        if( in.width() != out.width() || in.height() != out.height() || out.alignment() != 1u ||
            in.colorCount() != out.colorCount())
            throw imageException( "Bad input parameters in image function" );

        const uint32_t size = out.rowSize() * out.height();

        hipError_t error = hipMemcpy( out.data(), in.data(), size, hipMemcpyHostToDevice );
        if( error != hipSuccess )
            throw imageException( "Cannot copy a memory from CUDA device" );
    }

    Image ConvertToGrayScale( const Image & in )
    {
        Image_Function::ParameterValidation( in );

        Image out( in.width(), in.height() );

        ConvertToGrayScale( in, out );

        return out;
    }

    void ConvertToGrayScale( const Image & in, Image & out )
    {
        Image_Function::ParameterValidation( in, out );
        Image_Function::VerifyGrayScaleImage( out );

        if( in.colorCount() == GRAY_SCALE ) {
            Copy( in, out );
            return;
        }

        const uint32_t size = out.width() * out.height();
        const KernelParameters kernel = getKernelParameters( size );

        convertToGrayScaleCuda<<<kernel.blocksPerGrid, kernel.threadsPerBlock>>>(in.data(), out.data(), size, in.width(), in.colorCount());

        ValidateLastError();
    }

    void Copy( const Image & in, Image & out )
    {
        Image_Function::ParameterValidation( in, out );

        out = in;
    }

    void Fill( Image & image, uint8_t value )
    {
        Image_Function::ParameterValidation( image );

        const uint32_t size = image.rowSize() * image.height();
        const KernelParameters kernel = getKernelParameters( size );

        fillCuda<<<kernel.blocksPerGrid, kernel.threadsPerBlock>>>(image.data(), value, size);
        
        ValidateLastError();
    }

    Image GammaCorrection( const Image & in, double a, double gamma )
    {
        Image_Function::ParameterValidation( in );

        Image out( in.width(), in.height() );

        GammaCorrection( in, out, a, gamma );

        return out;
    }

    void GammaCorrection( const Image & in, Image & out, double a, double gamma )
    {
        Image_Function::ParameterValidation( in, out );

        if( a < 0 || gamma < 0 )
            throw imageException( "Bad input parameters in image function" );

        // We precalculate all values and store them in lookup table
        std::vector < uint8_t > value( 256 );

        for( uint16_t i = 0; i < 256; ++i ) {
            double data = a * pow( i / 255.0, gamma ) * 255 + 0.5;

            if( data < 256 )
                value[i] = static_cast<uint8_t>(data);
            else
                value[i] = 255;
        }

        LookupTable( in, out, value );
    }

    Image Invert( const Image & in )
    {
        Image_Function::ParameterValidation( in );

        Image out( in.width(), in.height() );

        Invert( in, out );

        return out;
    }

    void Invert( const Image & in, Image & out )
    {
        Image_Function::ParameterValidation( in, out );

        const uint32_t size = out.rowSize() * out.height();
        const KernelParameters kernel = getKernelParameters( size );

        invertCuda<<<kernel.blocksPerGrid, kernel.threadsPerBlock>>>(in.data(), out.data(), size);
        
        ValidateLastError();
    }

    Image LookupTable( const Image & in, const std::vector < uint8_t > & table )
    {
        Image_Function::ParameterValidation( in );

        Image out( in.width(), in.height() );

        LookupTable( in, out, table );

        return out;
    }
    
    void  LookupTable( const Image & in, Image & out, const std::vector < uint8_t > & table )
    {
        Image_Function::ParameterValidation( in, out );
        Image_Function::VerifyGrayScaleImage( in, out );

        if( table.size() != 256u )
            throw imageException( "Lookup table size is not equal to 256" );

        Cuda_Types::Array< uint8_t > tableCuda( table );

        const uint32_t size = out.rowSize() * out.height();
        const KernelParameters kernel = getKernelParameters( size );

        lookupTableCuda<<<kernel.blocksPerGrid, kernel.threadsPerBlock>>>(in.data(), out.data(), size, &tableCuda);

        ValidateLastError();
    }

    Image Maximum( const Image & in1, const Image & in2 )
    {
        Image_Function::ParameterValidation( in1, in2 );

        Image out( in1.width(), in1.height() );

        Maximum( in1, in2, out );

        return out;
    }

    void Maximum( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function::ParameterValidation( in1, in2, out );

        const uint32_t size = out.rowSize() * out.height();
        const KernelParameters kernel = getKernelParameters( size );

        maximumCuda<<<kernel.blocksPerGrid, kernel.threadsPerBlock>>>(in1.data(), in2.data(), out.data(), size);
        
        ValidateLastError();
    }

    Image Minimum( const Image & in1, const Image & in2 )
    {
        Image_Function::ParameterValidation( in1, in2 );

        Image out( in1.width(), in1.height() );

        Minimum( in1, in2, out );

        return out;
    }

    void Minimum( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function::ParameterValidation( in1, in2, out );

        const uint32_t size = out.rowSize() * out.height();
        const KernelParameters kernel = getKernelParameters( size );

        minimumCuda<<<kernel.blocksPerGrid, kernel.threadsPerBlock>>>(in1.data(), in2.data(), out.data(), size);
        
        ValidateLastError();
    }

    Image Subtract( const Image & in1, const Image & in2 )
    {
        Image_Function::ParameterValidation( in1, in2 );

        Image out( in1.width(), in1.height() );

        Subtract( in1, in2, out );

        return out;
    }

    void Subtract( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function::ParameterValidation( in1, in2, out );

        const uint32_t size = out.rowSize() * out.height();
        const KernelParameters kernel = getKernelParameters( size );

        subtractCuda<<<kernel.blocksPerGrid, kernel.threadsPerBlock>>>(in1.data(), in2.data(), out.data(), size);
        
        ValidateLastError();
    }

    Image Threshold( const Image & in, uint8_t threshold )
    {
        Image_Function::ParameterValidation( in );

        Image out( in.width(), in.height() );

        Threshold( in, out, threshold );

        return out;
    }

    void Threshold( const Image & in, Image & out, uint8_t threshold )
    {
        Image_Function::ParameterValidation( in, out );
        Image_Function::VerifyGrayScaleImage( in, out );

        const uint32_t size = out.width() * out.height();
        const KernelParameters kernel = getKernelParameters( size );

        thresholdCuda<<<kernel.blocksPerGrid, kernel.threadsPerBlock>>>(in.data(), out.data(), size, threshold);
        
        ValidateLastError();
    }

    Image Threshold( const Image & in, uint8_t minThreshold, uint8_t maxThreshold )
    {
        Image_Function::ParameterValidation( in );

        Image out( in.width(), in.height() );

        Threshold( in, out, minThreshold, maxThreshold );

        return out;
    }

    void Threshold( const Image & in, Image & out, uint8_t minThreshold, uint8_t maxThreshold )
    {
        Image_Function::ParameterValidation( in, out );
        Image_Function::VerifyGrayScaleImage( in, out );

        const uint32_t size = out.width() * out.height();
        const KernelParameters kernel = getKernelParameters( size );

        thresholdCuda<<<kernel.blocksPerGrid, kernel.threadsPerBlock>>>(in.data(), out.data(), size, minThreshold, maxThreshold);
        
        ValidateLastError();
    }
};
