#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../image_function.h"
#include "image_function_cuda.cuh"

namespace
{
	// Helper function which should return proper arguments for CUDA device functions
	void getKernelParameters(int & threadsPerBlock, int & blocksPerGrid, uint32_t size)
	{
		if( size < 256 ) {
			threadsPerBlock = size;
			blocksPerGrid = 1;
		}
		else {
			threadsPerBlock = 256;
			blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
		}
	};

	// The list of CUDA device functions
	__global__ void bitwiseAnd(const uint8_t * in1, const uint8_t * in2, uint8_t * out, uint32_t size)
	{
	    uint32_t i = blockDim.x * blockIdx.x + threadIdx.x;
	
	    if (i < size) {
	        out[i] = in1[i] & in2[i];
	    }
	};

	__global__ void bitwiseOr(const uint8_t * in1, const uint8_t * in2, uint8_t * out, uint32_t size)
	{
	    uint32_t i = blockDim.x * blockIdx.x + threadIdx.x;
	
	    if (i < size) {
	        out[i] = in1[i] | in2[i];
	    }
	};

	__global__ void bitwiseXor(const uint8_t * in1, const uint8_t * in2, uint8_t * out, uint32_t size)
	{
	    uint32_t i = blockDim.x * blockIdx.x + threadIdx.x;
	
	    if (i < size) {
	        out[i] = in1[i] ^ in2[i];
	    }
	};

	__global__ void invert(const uint8_t * in, uint8_t * out, uint32_t size)
	{
	    uint32_t i = blockDim.x * blockIdx.x + threadIdx.x;
	
	    if (i < size) {
	        out[i] = ~in[i];
	    }
	};
};

namespace Image_Function_Cuda
{
	template <uint8_t bytes>
	void ParameterValidation( const BitmapImageCuda <bytes> & image1 )
	{
		if( image1.empty() )
			throw imageException("Bad input parameters in image function");
	}

	template <uint8_t bytes1, uint8_t bytes2>
	void ParameterValidation( const BitmapImageCuda <bytes1> & image1, const BitmapImageCuda <bytes2> & image2 )
	{
		if( image1.empty() || image2.empty() || image1.width() != image2.width() || image1.height() != image2.height() )
			throw imageException("Bad input parameters in image function");
	}

	template <uint8_t bytes1, uint8_t bytes2, uint8_t bytes3>
	void ParameterValidation( const BitmapImageCuda <bytes1> & image1, const BitmapImageCuda <bytes2> & image2, const BitmapImageCuda <bytes3> & image3 )
	{
		if( image1.empty() || image2.empty() || image3.empty() || image1.width() != image2.width() || image1.height() != image2.height() ||
			image1.width() != image3.width() || image1.height() != image3.height() )
			throw imageException("Bad input parameters in image function");
	}


	ImageCuda BitwiseAnd( const ImageCuda & in1, const ImageCuda & in2 )
	{
		ParameterValidation( in1, in2 );

		ImageCuda out( in1.width(), in1.height() );

		BitwiseAnd( in1, in2, out );

		return out;
	}

	void BitwiseAnd( const ImageCuda & in1, const ImageCuda & in2, ImageCuda & out )
	{
		ParameterValidation( in1, in2, out );

		int threadsPerBlock = 1, blocksPerGrid = 1;
		getKernelParameters( threadsPerBlock, blocksPerGrid, out.width() * out.height() );

		bitwiseAnd<<<blocksPerGrid, threadsPerBlock>>>( in1.data(), in2.data(), out.data(), out.width() * out.height() );
		hipError_t error = hipGetLastError();
		if(error != hipSuccess)
			throw imageException("Failed to launch CUDA kernel");
	}

	ImageCuda BitwiseOr( const ImageCuda & in1, const ImageCuda & in2 )
	{
		ParameterValidation( in1, in2 );

		ImageCuda out( in1.width(), in1.height() );

		BitwiseOr( in1, in2, out );

		return out;
	}

	void BitwiseOr( const ImageCuda & in1, const ImageCuda & in2, ImageCuda & out )
	{
		ParameterValidation( in1, in2, out );

		int threadsPerBlock = 1, blocksPerGrid = 1;
		getKernelParameters( threadsPerBlock, blocksPerGrid, out.width() * out.height() );

		bitwiseOr<<<blocksPerGrid, threadsPerBlock>>>( in1.data(), in2.data(), out.data(), out.width() * out.height() );
		hipError_t error = hipGetLastError();
		if(error != hipSuccess)
			throw imageException("Failed to launch CUDA kernel");
	}

	ImageCuda BitwiseXor( const ImageCuda & in1, const ImageCuda & in2 )
	{
		ParameterValidation( in1, in2 );

		ImageCuda out( in1.width(), in1.height() );

		BitwiseXor( in1, in2, out );

		return out;
	}

	void BitwiseXor( const ImageCuda & in1, const ImageCuda & in2, ImageCuda & out )
	{
		ParameterValidation( in1, in2, out );

		int threadsPerBlock = 1, blocksPerGrid = 1;
		getKernelParameters( threadsPerBlock, blocksPerGrid, out.width() * out.height() );

		bitwiseXor<<<blocksPerGrid, threadsPerBlock>>>( in1.data(), in2.data(), out.data(), out.width() * out.height() );
		hipError_t error = hipGetLastError();
		if(error != hipSuccess)
			throw imageException("Failed to launch CUDA kernel");
	}

	void Convert( const Bitmap_Image::Image & in, ImageCuda & out )
	{
		Image_Function::ParameterValidation( in );
		ParameterValidation( out );

		if( in.width() != out.width() || in.height() != out.height() )
			throw imageException("Bad input parameters in image function");

		uint32_t rowSizeIn  = in.rowSize();
		uint32_t rowSizeOut = out.width();

		const uint8_t * Y    = in.data();
		const uint8_t * YEnd = Y + in.height() * rowSizeIn;

		uint8_t * cudaY = out.data();

		for( ; Y != YEnd; Y += rowSizeIn, cudaY += rowSizeOut ) {
			hipError_t error = hipMemcpy(cudaY, Y, out.width() * sizeof(uint8_t), hipMemcpyHostToDevice);
			if( error != hipSuccess )
				throw imageException("Cannot copy a memory to CUDA device");
		}
	}

	void Convert( const ImageCuda & in, Bitmap_Image::Image & out )
	{
		ParameterValidation( in );
		Image_Function::ParameterValidation( out );

		if( in.width() != out.width() || in.height() != out.height() )
			throw imageException("Bad input parameters in image function");

		uint32_t rowSizeIn  = in.width();
		uint32_t rowSizeOut = out.rowSize();

		      uint8_t * Y    = out.data();
		const uint8_t * YEnd = Y + out.height() * rowSizeOut;

		const uint8_t * cudaY = in.data();

		for( ; Y != YEnd; Y += rowSizeOut, cudaY += rowSizeIn ) {
			hipError_t error = hipMemcpy(Y, cudaY, in.width() * sizeof(uint8_t), hipMemcpyDeviceToHost);
			if( error != hipSuccess )
				throw imageException("Cannot copy a memory from CUDA device");
		}
	}

	ImageCuda Invert( const ImageCuda & in )
	{
		ParameterValidation( in );

		ImageCuda out( in.width(), in.height() );

		Invert( in, out );

		return out;
	}

	void Invert( const ImageCuda & in, ImageCuda & out )
	{
		ParameterValidation( in, out );

		int threadsPerBlock = 1, blocksPerGrid = 1;
		getKernelParameters( threadsPerBlock, blocksPerGrid, out.width() * out.height() );

		invert<<<blocksPerGrid, threadsPerBlock>>>( in.data(), out.data(), out.width() * out.height() );
		hipError_t error = hipGetLastError();
		if(error != hipSuccess)
			throw imageException("Failed to launch CUDA kernel");
	}
};
