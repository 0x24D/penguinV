#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../../Library/image_buffer.h"
#include "../../Library/image_function.h"
#include "../../Library/cuda/cuda_types.cuh"
#include "../../Library/cuda/image_function_cuda.cuh"
#include "unit_test_helper_cuda.cuh"

namespace
{
    // This function must run with thread count as 1
    __global__ void isEqualCuda( const uint8_t * image, uint8_t value, uint32_t size, uint32_t * differenceCount )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size )
        {
            if( image[id] != value )
                atomicAdd( differenceCount, 1 );
        }
    };

    __global__ void isAnyEqualCuda( const uint8_t * image, uint8_t * value, uint32_t valueCount, uint32_t size, uint32_t * differenceCount )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( id < size )
        {
            bool equal = false;

            for( uint32_t i = 0; i < valueCount; ++i )
            {
                if( image[id] == value[i] )
                {
                    equal = true;
                    break;
                }
            }

            if( !equal )
                atomicAdd( differenceCount, 1 );
        }
    };
};

namespace Unit_Test
{
    namespace Cuda
    {
        Bitmap_Image_Cuda::Image uniformImage( uint8_t value )
        {
            Bitmap_Image_Cuda::Image image( randomValue<uint32_t>( 1, 2048 ), randomValue<uint32_t>( 1, 2048 ) );

            image.fill( value );

            return image;
        }

        Bitmap_Image_Cuda::Image uniformImage()
        {
            return uniformImage( randomValue<uint8_t>( 256 ) );
        }

        Bitmap_Image_Cuda::Image uniformColorImage()
        {
            return uniformColorImage( randomValue<uint8_t>( 256 ) );
        }

        Bitmap_Image_Cuda::Image uniformColorImage( uint8_t value )
        {
            Bitmap_Image_Cuda::Image image( randomValue<uint32_t>( 1, 2048 ), randomValue<uint32_t>( 1, 2048 ), Bitmap_Image_Cuda::RGB );

            image.fill( value );

            return image;
        }

        Bitmap_Image_Cuda::Image blackImage()
        {
            return uniformImage( 0u );
        }

        Bitmap_Image_Cuda::Image whiteImage()
        {
            return uniformImage( 255u );
        }

        std::vector < Bitmap_Image_Cuda::Image > uniformImages( uint32_t images )
        {
            if( images == 0 )
                throw imageException( "Invalid parameter" );

            std::vector < Bitmap_Image_Cuda::Image > image;

            image.push_back( uniformImage() );

            image.resize( images );

            for( size_t i = 1; i < image.size(); ++i ) {
                image[i].resize( image[0].width(), image[0].height() );
                image[i].fill( randomValue<uint8_t>( 256 ) );
            }

            return image;
        }

        std::vector < Bitmap_Image_Cuda::Image > uniformImages( std::vector < uint8_t > intensityValue )
        {
            if( intensityValue.size() == 0 )
                throw imageException( "Invalid parameter" );

            std::vector < Bitmap_Image_Cuda::Image > image;

            image.push_back( uniformImage( intensityValue[0] ) );

            image.resize( intensityValue.size() );

            for( size_t i = 1; i < image.size(); ++i ) {
                image[i].resize( image[0].width(), image[0].height() );
                image[i].fill( intensityValue[i] );
            }

            return image;
        }

        bool verifyImage( const Bitmap_Image_Cuda::Image & image, uint8_t value )
        {
            Cuda_Types::_cuint32_t differenceCount( 0 );

            const uint32_t size = image.rowSize() * image.height();

            isEqualCuda<<< (size + 255) / 256, 256 >>>(image.data(), value, size, &differenceCount);
            hipError_t error = hipGetLastError();
            if( error != hipSuccess )
                throw imageException( "Failed to launch CUDA kernel" );

            return differenceCount.get() == 0;
        }

        bool verifyImage( const Bitmap_Image_Cuda::Image & image, const std::vector < uint8_t > & value )
        {
            Cuda_Types::_cuint32_t differenceCount( 0 );
            Cuda_Types::Array<uint8_t> valueCuda( value );

            const uint32_t size = image.rowSize() * image.height();

            isAnyEqualCuda<<< (size + 255) / 256, 256 >>>(image.data(), &valueCuda, valueCuda.size(), size, &differenceCount);
            hipError_t error = hipGetLastError();
            if( error != hipSuccess )
                throw imageException( "Failed to launch CUDA kernel" );

            return differenceCount.get() == 0;
        }
    };
};
