#include "hip/hip_runtime.h"
#include "../../Library/image_buffer.h"
#include "../../Library/image_function.h"
#include "../../Library/cuda/cuda_types.cuh"
#include "../../Library/cuda/image_function_cuda.cuh"
#include "unit_test_helper_cuda.cuh"

namespace
{
    // This function must run with thread count as 1
    __global__ void isEqual( const uint8_t * image, uint8_t value, uint32_t size, uint8_t * equal )
    {
        uint32_t id = blockDim.x * blockIdx.x + threadIdx.x;

        if( image[id] != value )
            *equal = false;
    };
};

namespace Unit_Test
{
    namespace Cuda
    {
        Bitmap_Image_Cuda::Image uniformImage( uint8_t value )
        {
            Bitmap_Image_Cuda::Image image( randomValue<uint32_t>( 1, 2048 ), randomValue<uint32_t>( 1, 2048 ) );

            image.fill( value );

            return image;
        }

        Bitmap_Image_Cuda::Image uniformImage()
        {
            return uniformImage( randomValue<uint8_t>( 256 ) );
        }

        Bitmap_Image_Cuda::Image blackImage()
        {
            return uniformImage( 0u );
        }

        Bitmap_Image_Cuda::Image whiteImage()
        {
            return uniformImage( 255u );
        }

        std::vector < Bitmap_Image_Cuda::Image > uniformImages( uint32_t images )
        {
            if( images == 0 )
                throw imageException( "Invalid parameter" );

            std::vector < Bitmap_Image_Cuda::Image > image;

            image.push_back( uniformImage() );

            image.resize( images );

            for( size_t i = 1; i < image.size(); ++i ) {
                image[i].resize( image[0].width(), image[0].height() );
                image[i].fill( randomValue<uint8_t>( 256 ) );
            }

            return image;
        }

        std::vector < Bitmap_Image_Cuda::Image > uniformImages( std::vector < uint8_t > intensityValue )
        {
            if( intensityValue.size() == 0 )
                throw imageException( "Invalid parameter" );

            std::vector < Bitmap_Image_Cuda::Image > image;

            image.push_back( uniformImage( intensityValue[0] ) );

            image.resize( intensityValue.size() );

            for( size_t i = 1; i < image.size(); ++i ) {
                image[i].resize( image[0].width(), image[0].height() );
                image[i].fill( intensityValue[i] );
            }

            return image;
        }

        bool verifyImage( const Bitmap_Image_Cuda::Image & image, uint8_t value )
        {
            Cuda_Types::_cbool equal( true );

            isEqual<<< max( image.width(), image.height() ), 1 >>>(image.data(), value, image.width() * image.height(), &equal);
            hipError_t error = hipGetLastError();
            if( error != hipSuccess )
                throw imageException( "Failed to launch CUDA kernel" );

            return equal.get() != 0;
        }
    };
};
