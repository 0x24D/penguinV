#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include "image_function_cuda.cuh"
#include "../parameter_validation.h"
#include "../thirdparty/multicuda/src/cuda_types.cuh"
#include "../thirdparty/multicuda/src/cuda_helper.cuh"

namespace
{
    // The list of CUDA device functions on device side
    __global__ void absoluteDifferenceCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2,
                                            uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint8_t * in1X = in1 + y * rowSizeIn1 + x;
            const uint8_t * in2X = in2 + y * rowSizeIn2 + x;
            uint8_t * outX = out + y * rowSizeOut + x;
            (*outX) = ((*in1X) > ( *in2X )) ? ((*in1X) - (*in2X)) : ((*in2X) - (*in1X));
        }
    }

    __global__ void bitwiseAndCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2,
                                    uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint32_t idIn1 = y * rowSizeIn1 + x;
            const uint32_t idIn2 = y * rowSizeIn2 + x;
            const uint32_t idOut = y * rowSizeOut + x;
            out[idOut] = in1[idIn1] & in2[idIn2];
        }
    }

    __global__ void bitwiseOrCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2,
                                   uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint32_t idIn1 = y * rowSizeIn1 + x;
            const uint32_t idIn2 = y * rowSizeIn2 + x;
            const uint32_t idOut = y * rowSizeOut + x;
            out[idOut] = in1[idIn1] | in2[idIn2];
        }
    }

    __global__ void bitwiseXorCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2,
                                    uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint32_t idIn1 = y * rowSizeIn1 + x;
            const uint32_t idIn2 = y * rowSizeIn2 + x;
            const uint32_t idOut = y * rowSizeOut + x;
            out[idOut] = in1[idIn1] ^ in2[idIn2];
        }
    }

    __global__ void convertToGrayScaleCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t colorCount, uint8_t * out, uint32_t rowSizeOut,
                                            uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint8_t * data = in + y * rowSizeIn + x * colorCount;
            const uint8_t * dataEnd = data + colorCount;

            uint32_t sum = 0;
            for ( ; data != dataEnd; ++data )
            {
                sum += (*data);
            }

            const uint32_t id = y * rowSizeOut + x;
            out[id] = static_cast<uint8_t>(sum / colorCount);
        }
    }

    __global__ void convertToRgbCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint8_t colorCount,
                                      uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint8_t * dataIn = in + y * rowSizeIn + x;

            uint8_t * dataOut = out + (width * y + x) * colorCount;
            const uint8_t * dataOutEnd = dataOut + colorCount;

            for ( ; dataOut != dataOutEnd; ++dataOut )
            {
                (*dataOut) = (*dataIn);
            }
        }
    }

    __global__ void copyCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            out[y * rowSizeOut + x] = in[y * rowSizeIn + x];
        }
    }

    __global__ void extractChannelCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t colorCount, uint8_t * out, uint32_t rowSizeOut,
                                        uint32_t width, uint32_t height, uint8_t channelId )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height )
            out[y * rowSizeOut + x] = in[y * rowSizeIn + x * colorCount + channelId];
    }

    __global__ void fillCuda( uint8_t * data, uint32_t rowSize, uint32_t width, uint32_t height, uint8_t value )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height )
            data[y * rowSize + x] = value;
    }

    __global__ void flipCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height,
                              bool horizontal, bool vertical )
    {
        const uint32_t inX = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t inY = blockDim.y * blockIdx.y + threadIdx.y;

        if ( inX < width && inY < height ) {
            const uint32_t outX = horizontal ? (width  - 1 - inX) : inX;
            const uint32_t outY = vertical   ? (height - 1 - inY) : inY;

            out[outY * rowSizeOut + outX] = in[inY * rowSizeIn + inX];
        }
    }

    __global__ void histogramCuda( const uint8_t * data, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t * histogram )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint32_t id = y * rowSize + x;
            atomicAdd( &histogram[data[id]], 1 );
        }
    }

    __global__ void invertCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            out[y * rowSizeOut + x] = ~in[y * rowSizeIn + x];
        }
    }

    __global__ void lookupTableCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut,
                                     uint32_t width, uint32_t height, uint8_t * table )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            out[y * rowSizeOut + x] = table[in[y * rowSizeIn + x]];
        }
    }

    __global__ void maximumCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2,
                                 uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint8_t * in1X = in1 + y * rowSizeIn1 + x;
            const uint8_t * in2X = in2 + y * rowSizeIn2 + x;
            uint8_t * outX = out + y * rowSizeOut + x;
            (*outX) = ((*in1X) > ( *in2X )) ? (*in1X) : (*in2X);
        }
    }

    __global__ void minimumCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2,
                                 uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint8_t * in1X = in1 + y * rowSizeIn1 + x;
            const uint8_t * in2X = in2 + y * rowSizeIn2 + x;
            uint8_t * outX = out + y * rowSizeOut + x;
            (*outX) = ((*in1X) < (*in2X)) ? (*in1X) : (*in2X);
        }
    }

    __global__ void subtractCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2,
                                  uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint8_t * in1X = in1 + y * rowSizeIn1 + x;
            const uint8_t * in2X = in2 + y * rowSizeIn2 + x;
            uint8_t * outX = out + y * rowSizeOut + x;
            (*outX) = ((*in1X) > ( *in2X )) ? ((*in1X) - (*in2X)) : 0;
        }
    }

    __global__ void thresholdCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height,
                                   uint8_t threshold )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            out[y * rowSizeOut + x] = (in[y * rowSizeIn + x] < threshold) ? 0 : 255;
        }
    }

    __global__ void thresholdCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height,
                                   uint8_t minThreshold, uint8_t maxThreshold )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint32_t idIn = y * rowSizeIn + x;
            out[y * rowSizeOut + x] = ((in[idIn] < minThreshold) || (in[idIn] > maxThreshold)) ? 0 : 255;
        }
    }
}

namespace Image_Function_Cuda
{
    Image AbsoluteDifference( const Image & in1, const Image & in2 )
    {
        Image_Function::ParameterValidation( in1, in2 );

        Image out( in1.width(), in1.height() );

        AbsoluteDifference( in1, 0, 0, in2, 0, 0, out, 0, 0, out.width(), out.height() );

        return out;
    }

    void AbsoluteDifference( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function::ParameterValidation( in1, in2, out );

        AbsoluteDifference( in1, 0, 0, in2, 0, 0, out, 0, 0, out.width(), out.height() );
    }

    Image AbsoluteDifference( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                              uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in1, startX1, startY1, in2, startX2, startY2, width, height );

        Image out( width, height );

        AbsoluteDifference( in1, startX1, startY1, in2, startX2, startY2, out, 0, 0, out.width(), out.height() );

        return out;
    }

    void AbsoluteDifference( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                             Image & out, uint32_t startXOut, uint32_t startYOut, uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in1, startX1, startY1, in2, startX2, startY2, out, startXOut, startYOut, width, height );

        const uint8_t colorCount = Image_Function::CommonColorCount( in1, in2, out );
        width = width * colorCount;

        const uint32_t rowSizeIn1 = in1.rowSize();
        const uint32_t rowSizeIn2 = in2.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t * in1Y = in1.data() + startY1   * rowSizeIn1 + startX1   * colorCount;
        const uint8_t * in2Y = in2.data() + startY2   * rowSizeIn2 + startX2   * colorCount;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut * colorCount;

        launchKernel2D( absoluteDifferenceCuda, width, height,
                        in1Y, rowSizeIn1, in2Y, rowSizeIn2, outY, rowSizeOut, width, height );
    }

    Image BitwiseAnd( const Image & in1, const Image & in2 )
    {
        Image_Function::ParameterValidation( in1, in2 );

        Image out( in1.width(), in1.height() );

        BitwiseAnd( in1, 0, 0, in2, 0, 0, out, 0, 0, out.width(), out.height() );

        return out;
    }

    void BitwiseAnd( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function::ParameterValidation( in1, in2, out );

        BitwiseAnd( in1, 0, 0, in2, 0, 0, out, 0, 0, out.width(), out.height() );
    }

    Image BitwiseAnd( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                      uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in1, startX1, startY1, in2, startX2, startY2, width, height );

        Image out( width, height );

        BitwiseAnd( in1, startX1, startY1, in2, startX2, startY2, out, 0, 0, out.width(), out.height() );

        return out;
    }

    void BitwiseAnd( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                     Image & out, uint32_t startXOut, uint32_t startYOut, uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in1, startX1, startY1, in2, startX2, startY2, out, startXOut, startYOut, width, height );

        const uint8_t colorCount = Image_Function::CommonColorCount( in1, in2, out );
        width = width * colorCount;

        const uint32_t rowSizeIn1 = in1.rowSize();
        const uint32_t rowSizeIn2 = in2.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t * in1Y = in1.data() + startY1   * rowSizeIn1 + startX1   * colorCount;
        const uint8_t * in2Y = in2.data() + startY2   * rowSizeIn2 + startX2   * colorCount;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut * colorCount;

        launchKernel2D( bitwiseAndCuda, width, height,
                        in1Y, rowSizeIn1, in2Y, rowSizeIn2, outY, rowSizeOut, width, height );
    }

    Image BitwiseOr( const Image & in1, const Image & in2 )
    {
        Image_Function::ParameterValidation( in1, in2 );

        Image out( in1.width(), in1.height() );

        BitwiseOr( in1, 0, 0, in2, 0, 0, out, 0, 0, out.width(), out.height() );

        return out;
    }

    void BitwiseOr( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function::ParameterValidation( in1, in2, out );

        BitwiseOr( in1, 0, 0, in2, 0, 0, out, 0, 0, out.width(), out.height() );
    }

    Image BitwiseOr( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                     uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in1, startX1, startY1, in2, startX2, startY2, width, height );

        Image out( width, height );

        BitwiseOr( in1, startX1, startY1, in2, startX2, startY2, out, 0, 0, out.width(), out.height() );

        return out;
    }

    void BitwiseOr( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                    Image & out, uint32_t startXOut, uint32_t startYOut, uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in1, startX1, startY1, in2, startX2, startY2, out, startXOut, startYOut, width, height );

        const uint8_t colorCount = Image_Function::CommonColorCount( in1, in2, out );
        width = width * colorCount;

        const uint32_t rowSizeIn1 = in1.rowSize();
        const uint32_t rowSizeIn2 = in2.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t * in1Y = in1.data() + startY1   * rowSizeIn1 + startX1   * colorCount;
        const uint8_t * in2Y = in2.data() + startY2   * rowSizeIn2 + startX2   * colorCount;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut * colorCount;

        launchKernel2D( bitwiseOrCuda, width, height,
                        in1Y, rowSizeIn1, in2Y, rowSizeIn2, outY, rowSizeOut, width, height );
    }

    Image BitwiseXor( const Image & in1, const Image & in2 )
    {
        Image_Function::ParameterValidation( in1, in2 );

        Image out( in1.width(), in1.height() );

        BitwiseXor( in1, 0, 0, in2, 0, 0, out, 0, 0, out.width(), out.height() );

        return out;
    }

    void BitwiseXor( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function::ParameterValidation( in1, in2, out );

        BitwiseXor( in1, 0, 0, in2, 0, 0, out, 0, 0, out.width(), out.height() );
    }

    Image BitwiseXor( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                      uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in1, startX1, startY1, in2, startX2, startY2, width, height );

        Image out( width, height );

        BitwiseXor( in1, startX1, startY1, in2, startX2, startY2, out, 0, 0, out.width(), out.height() );

        return out;
    }

    void BitwiseXor( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                     Image & out, uint32_t startXOut, uint32_t startYOut, uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in1, startX1, startY1, in2, startX2, startY2, out, startXOut, startYOut, width, height );

        const uint8_t colorCount = Image_Function::CommonColorCount( in1, in2, out );
        width = width * colorCount;

        const uint32_t rowSizeIn1 = in1.rowSize();
        const uint32_t rowSizeIn2 = in2.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t * in1Y = in1.data() + startY1   * rowSizeIn1 + startX1   * colorCount;
        const uint8_t * in2Y = in2.data() + startY2   * rowSizeIn2 + startX2   * colorCount;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut * colorCount;

        launchKernel2D( bitwiseXorCuda, width, height,
                        in1Y, rowSizeIn1, in2Y, rowSizeIn2, outY, rowSizeOut, width, height );
    }

    Image ConvertToCuda( const PenguinV_Image::Image & in )
    {
        Image out( in.width(), in.height(), in.colorCount() );

        ConvertToCuda( in, out );

        return out;
    }

    void ConvertToCuda( const PenguinV_Image::Image & in, Image & out )
    {
        Image_Function::ParameterValidation( in );
        Image_Function::ParameterValidation( out );

        if ( in.width() != out.width() || in.height() != out.height() ||
             in.colorCount() != out.colorCount() )
            throw imageException( "Bad input parameters in image function" );

        if ( in.alignment() == 1u || (in.rowSize() == in.width() * in.colorCount()) )
        {
            const uint32_t size = in.rowSize() * in.height();

            if ( !multiCuda::cudaSafeCheck( hipMemcpy( out.data(), in.data(), size * sizeof( uint8_t ), hipMemcpyHostToDevice ) ) )
                throw imageException( "Cannot copy a memory to CUDA device" );
        }
        else
        {
            if ( !multiCuda::cudaSafeCheck( hipMemcpy2D( out.data(), out.rowSize(), in.data(), in.rowSize(),
                                                          in.colorCount() * in.width(), in.height(), hipMemcpyHostToDevice ) ) )
                throw imageException( "Cannot copy a memory to CUDA device" );
        }
    }

    PenguinV_Image::Image ConvertFromCuda( const Image & in )
    {
        PenguinV_Image::Image out( in.width(), in.height(), in.colorCount(), 1u );

        ConvertFromCuda( in, out );

        return out;
    }

    void ConvertFromCuda( const Image & in, PenguinV_Image::Image & out )
    {
        Image_Function::ParameterValidation( in );
        Image_Function::ParameterValidation( out );

        if ( in.width() != out.width() || in.height() != out.height() ||
             in.colorCount() != out.colorCount() )
            throw imageException( "Bad input parameters in image function" );

        if ( out.alignment() == 1u || (out.rowSize() == out.width() * out.colorCount()) )
        {
            const uint32_t size = in.rowSize() * in.height();

            if ( !multiCuda::cudaSafeCheck( hipMemcpy( out.data(), in.data(), size, hipMemcpyDeviceToHost ) ) )
                throw imageException( "Cannot copy a memory from CUDA device" );
        }
        else
        {
            if ( !multiCuda::cudaSafeCheck( hipMemcpy2D( out.data(), out.rowSize(), in.data(), in.rowSize(),
                                                          in.colorCount() * in.width(), in.height(), hipMemcpyDeviceToHost ) ) )
                throw imageException( "Cannot copy a memory to CUDA device" );
        }
    }

    Image ConvertToGrayScale( const Image & in )
    {
        Image_Function::ParameterValidation( in );

        Image out( in.width(), in.height() );

        ConvertToGrayScale( in, 0, 0, out, 0, 0, out.width(), out.height() );

        return out;
    }

    void ConvertToGrayScale( const Image & in, Image & out )
    {
        Image_Function::ParameterValidation( in, out );

        ConvertToGrayScale( in, 0, 0, out, 0, 0, out.width(), out.height() );
    }

    Image ConvertToGrayScale( const Image & in, uint32_t startXIn, uint32_t startYIn, uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, width, height );

        Image out( width, height );

        ConvertToGrayScale( in, startXIn, startYIn, out, 0, 0, width, height );

        return out;
    }

    void ConvertToGrayScale( const Image & in, uint32_t startXIn, uint32_t startYIn, Image & out, uint32_t startXOut, uint32_t startYOut,
                             uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, out, startXOut, startYOut, width, height );
        Image_Function::VerifyGrayScaleImage( out );

        if ( in.colorCount() == GRAY_SCALE ) {
            Copy( in, startXIn, startYIn, out, startXOut, startYOut, width, height );
            return;
        }

        const uint32_t rowSizeIn  = in.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t colorCount = in.colorCount();

        const uint8_t * inY  = in.data()  + startYIn  * rowSizeIn  + startXIn * colorCount;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut;

        launchKernel2D( convertToGrayScaleCuda, width, height,
                        inY, rowSizeIn, colorCount, outY, rowSizeOut, width, height );
    }

    Image ConvertToRgb( const Image & in )
    {
        Image_Function::ParameterValidation( in );

        Image out( in.width(), in.height(), RGB );

        ConvertToRgb( in, 0, 0, out, 0, 0, out.width(), out.height() );

        return out;
    }

    void ConvertToRgb( const Image & in, Image & out )
    {
        Image_Function::ParameterValidation( in, out );

        ConvertToRgb( in, 0, 0, out, 0, 0, out.width(), out.height() );
    }

    Image ConvertToRgb( const Image & in, uint32_t startXIn, uint32_t startYIn, uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, width, height );

        Image out( width, height, RGB );

        ConvertToRgb( in, startXIn, startYIn, out, 0, 0, width, height );

        return out;
    }

    void ConvertToRgb( const Image & in, uint32_t startXIn, uint32_t startYIn, Image & out, uint32_t startXOut, uint32_t startYOut,
                       uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, out, startXOut, startYOut, width, height );
        Image_Function::VerifyColoredImage  ( out );

        if ( in.colorCount() == RGB ) {
            Copy( in, startXIn, startYIn, out, startXOut, startYOut, width, height );
            return;
        }

        const uint32_t rowSizeIn  = in.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t colorCount = out.colorCount();

        const uint8_t * inY  = in.data()  + startYIn  * rowSizeIn  + startXIn;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut * colorCount;

        launchKernel2D( convertToRgbCuda, width, height,
                        inY, rowSizeIn, outY, rowSizeOut, colorCount, width, height );
    }

    void Copy( const Image & in, Image & out )
    {
        Image_Function::ParameterValidation( in, out );

        out = in;
    }

    Image Copy( const Image & in, uint32_t startXIn, uint32_t startYIn, uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, width, height );

        Image out( width, height );

        Copy( in, startXIn, startYIn, out, 0, 0, width, height );

        return out;
    }

    void Copy( const Image & in, uint32_t startXIn, uint32_t startYIn, Image & out, uint32_t startXOut, uint32_t startYOut,
               uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, out, startXOut, startYOut, width, height );

        const uint8_t colorCount  = Image_Function::CommonColorCount( in, out );
        const uint32_t rowSizeIn  = in.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        width = width * colorCount;

        const uint8_t * inY  = in.data()  + startYIn  * rowSizeIn  + startXIn  * colorCount;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut * colorCount;

        launchKernel2D( copyCuda, width, height,
                        inY, rowSizeIn, outY, rowSizeOut, width, height );
    }

    Image ExtractChannel( const Image & in, uint8_t channelId )
    {
        Image_Function::ParameterValidation( in );

        Image out( in.width(), in.height() );

        ExtractChannel( in, 0, 0, out, 0, 0, in.width(), in.height(), channelId );

        return out;
    }

    void ExtractChannel( const Image & in, Image & out, uint8_t channelId )
    {
        Image_Function::ParameterValidation( in, out );

        ExtractChannel( in, 0, 0, out, 0, 0, in.width(), in.height(), channelId );
    }

    Image ExtractChannel( const Image & in, uint32_t x, uint32_t y, uint32_t width, uint32_t height, uint8_t channelId )
    {
        Image_Function::ParameterValidation( in, x, y, width, height );

        Image out( width, height );

        ExtractChannel( in, x, y, out, 0, 0, width, height, channelId );

        return out;
    }

    void ExtractChannel( const Image & in, uint32_t startXIn, uint32_t startYIn, Image & out, uint32_t startXOut,
                         uint32_t startYOut, uint32_t width, uint32_t height, uint8_t channelId )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, out, startXOut, startYOut, width, height );
        Image_Function::VerifyGrayScaleImage( out );

        if ( channelId >= in.colorCount() )
            throw imageException( "Channel ID for color image is greater than channel count in input image" );

        const uint32_t rowSizeIn  = in.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t colorCount = in.colorCount();

        const uint8_t * inY  = in.data()  + startYIn  * rowSizeIn  + startXIn * colorCount + channelId;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut;

        launchKernel2D( extractChannelCuda, width, height,
                        inY, rowSizeIn, colorCount, outY, rowSizeOut, width, height, channelId );
    }

    void Fill( Image & image, uint8_t value )
    {
        image.fill( value );
    }

    void Fill( Image & image, uint32_t x, uint32_t y, uint32_t width, uint32_t height, uint8_t value )
    {
        Image_Function::ParameterValidation( image, x, y, width, height );
        Image_Function::VerifyGrayScaleImage( image );

        const uint32_t rowSize = image.rowSize();

        uint8_t * imageY = image.data() + y * rowSize + x;

        launchKernel2D( fillCuda, width, height,
                        imageY, rowSize, width, height, value );
    }

    Image Flip( const Image & in, bool horizontal, bool vertical )
    {
        Image_Function::ParameterValidation( in );

        Image out( in.width(), in.height() );

        Flip( in, out, horizontal, vertical );

        return out;
    }

    void  Flip( const Image & in, Image & out, bool horizontal, bool vertical )
    {
        Image_Function::ParameterValidation( in, out );
        Image_Function::VerifyGrayScaleImage( in, out );

        if ( !horizontal && !vertical ) {
            Copy( in, out );
        }
        else {
            launchKernel2D( flipCuda, out.width(), out.height(),
                            in.data(), in.rowSize(), out.data(), out.rowSize(), out.width(), out.height(), horizontal, vertical );
        }
    }

    Image GammaCorrection( const Image & in, double a, double gamma )
    {
        Image_Function::ParameterValidation( in );

        Image out( in.width(), in.height() );

        GammaCorrection( in, 0, 0, out, 0, 0, out.width(), out.height(), a, gamma );

        return out;
    }

    void GammaCorrection( const Image & in, Image & out, double a, double gamma )
    {
        Image_Function::ParameterValidation( in, out );

        GammaCorrection( in, 0, 0, out, 0, 0, out.width(), out.height(), a, gamma );
    }

    Image GammaCorrection( const Image & in, uint32_t startXIn, uint32_t startYIn, uint32_t width, uint32_t height, double a, double gamma )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, width, height );

        Image out( width, height );

        GammaCorrection( in, startXIn, startYIn, out, 0, 0, width, height, a, gamma );

        return out;
    }

    void GammaCorrection( const Image & in, uint32_t startXIn, uint32_t startYIn, Image & out, uint32_t startXOut, uint32_t startYOut,
                          uint32_t width, uint32_t height, double a, double gamma )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, out, startXOut, startYOut, width, height );
        Image_Function::VerifyGrayScaleImage( in, out );

        if ( a < 0 || gamma < 0 )
            throw imageException( "Gamma correction parameters are invalid" );

        // We precalculate all values and store them in lookup table
        std::vector < uint8_t > value( 256, 255u );

        for ( uint16_t i = 0; i < 256; ++i ) {
            double data = a * pow( i / 255.0, gamma ) * 255 + 0.5;

            if ( data < 256 )
                value[i] = static_cast<uint8_t>(data);
        }

        LookupTable( in, startXIn, startYIn, out, startXOut, startYOut, width, height, value );
    }

    uint8_t GetThreshold( const std::vector < uint32_t > & histogram )
    {
        if ( histogram.size() != 256 )
            throw imageException( "Histogram size is not 256" );

        // It is well-known Otsu's method to find threshold
        uint32_t pixelCount = histogram[0] + histogram[1];
        uint32_t sum = histogram[1];
        for ( uint16_t i = 2; i < 256; ++i ) {
            sum = sum + i * histogram[i];
            pixelCount += histogram[i];
        }

        uint32_t sumTemp = 0;
        uint32_t pixelCountTemp = 0;

        double maximumSigma = -1;

        uint8_t threshold = 0;

        for ( uint16_t i = 0; i < 256; ++i ) {
            pixelCountTemp += histogram[i];

            if ( pixelCountTemp > 0 && pixelCountTemp != pixelCount ) {
                sumTemp += i * histogram[i];

                double w1 = static_cast<double>(pixelCountTemp) / pixelCount;
                double a  = static_cast<double>(sumTemp) / pixelCountTemp -
                    static_cast<double>(sum - sumTemp) / (pixelCount - pixelCountTemp);
                double sigma = w1 * (1 - w1) * a * a;

                if ( sigma > maximumSigma ) {
                    maximumSigma = sigma;
                    threshold = static_cast <uint8_t>(i);
                }
            }
        }

        return threshold;
    }

    std::vector < uint32_t > Histogram( const Image & image )
    {
        std::vector < uint32_t > histogram;

        Histogram( image, 0, 0, image.width(), image.height(), histogram );

        return histogram;
    }

    void Histogram( const Image & image, std::vector < uint32_t > & histogram )
    {
        Histogram( image, 0, 0, image.width(), image.height(), histogram );
    }

    std::vector < uint32_t > Histogram( const Image & image, uint32_t x, uint32_t y, uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( image, x, y, width, height );

        std::vector < uint32_t > histogram;

        Histogram( image, x, y, width, height, histogram );

        return histogram;
    }

    void Histogram( const Image & image, uint32_t x, uint32_t y, uint32_t width, uint32_t height, std::vector < uint32_t > & histogram )
    {
        Image_Function::ParameterValidation( image, x, y, width, height );
        Image_Function::VerifyGrayScaleImage( image );

        histogram.resize( 256u );
        std::fill( histogram.begin(), histogram.end(), 0u );

        const uint32_t rowSize = image.rowSize();

        const uint8_t * imageY = image.data() + y * rowSize + x;

        multiCuda::Array< uint32_t > tableCuda( histogram );

        launchKernel2D( histogramCuda, width, height,
                        imageY, rowSize, width, height, tableCuda.data() );

        histogram = tableCuda.get();
    }

    Image Invert( const Image & in )
    {
        Image_Function::ParameterValidation( in );

        Image out( in.width(), in.height() );

        Invert( in, 0, 0, out, 0, 0, out.width(), out.height() );

        return out;
    }

    void Invert( const Image & in, Image & out )
    {
        Image_Function::ParameterValidation( in, out );

        Invert( in, 0, 0, out, 0, 0, out.width(), out.height() );
    }

    Image Invert( const Image & in, uint32_t startXIn, uint32_t startYIn, uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, width, height );

        Image out( width, height );

        Invert( in, startXIn, startYIn, out, 0, 0, width, height );

        return out;
    }

    void Invert( const Image & in, uint32_t startXIn, uint32_t startYIn, Image & out, uint32_t startXOut, uint32_t startYOut,
                 uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, out, startXOut, startYOut, width, height );

        const uint8_t colorCount = Image_Function::CommonColorCount( in, out );
        width = width * colorCount;

        const uint32_t rowSizeIn  = in.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t * inY  = in.data()  + startYIn  * rowSizeIn  + startXIn  * colorCount;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut * colorCount;

        launchKernel2D( invertCuda, width, height,
                        inY, rowSizeIn, outY, rowSizeOut, width, height );
    }

    Image LookupTable( const Image & in, const std::vector < uint8_t > & table )
    {
        Image_Function::ParameterValidation( in );

        Image out( in.width(), in.height() );

        LookupTable( in, 0, 0, out, 0, 0, out.width(), out.height(), table );

        return out;
    }

    void LookupTable( const Image & in, Image & out, const std::vector < uint8_t > & table )
    {
        Image_Function::ParameterValidation( in, out );

        LookupTable( in, 0, 0, out, 0, 0, out.width(), out.height(), table );
    }

    Image LookupTable( const Image & in, uint32_t startXIn, uint32_t startYIn, uint32_t width, uint32_t height,
                       const std::vector < uint8_t > & table )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, width, height );

        Image out( width, height );

        LookupTable( in, startXIn, startYIn, out, 0, 0, width, height, table );

        return out;
    }

    void LookupTable( const Image & in, uint32_t startXIn, uint32_t startYIn, Image & out, uint32_t startXOut, uint32_t startYOut,
                      uint32_t width, uint32_t height, const std::vector < uint8_t > & table )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, out, startXOut, startYOut, width, height );
        Image_Function::VerifyGrayScaleImage( in, out );

        if ( table.size() != 256u )
            throw imageException( "Lookup table size is not equal to 256" );

        const uint32_t rowSizeIn  = in.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t * inY  = in.data()  + startYIn  * rowSizeIn  + startXIn;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut;

        multiCuda::Array< uint8_t > tableCuda( table );

        launchKernel2D( lookupTableCuda, width, height,
                        inY, rowSizeIn, outY, rowSizeOut, width, height, tableCuda.data() );
    }

    Image Maximum( const Image & in1, const Image & in2 )
    {
        Image_Function::ParameterValidation( in1, in2 );

        Image out( in1.width(), in1.height() );

        Maximum( in1, 0, 0, in2, 0, 0, out, 0, 0, out.width(), out.height() );

        return out;
    }

    void Maximum( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function::ParameterValidation( in1, in2, out );

        Maximum( in1, 0, 0, in2, 0, 0, out, 0, 0, out.width(), out.height() );
    }

    Image Maximum( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                   uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in1, startX1, startY1, in2, startX2, startY2, width, height );

        Image out( width, height );

        Maximum( in1, startX1, startY1, in2, startX2, startY2, out, 0, 0, out.width(), out.height() );

        return out;
    }

    void Maximum( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                  Image & out, uint32_t startXOut, uint32_t startYOut, uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in1, startX1, startY1, in2, startX2, startY2, out, startXOut, startYOut, width, height );

        const uint8_t colorCount = Image_Function::CommonColorCount( in1, in2, out );
        width = width * colorCount;

        const uint32_t rowSizeIn1 = in1.rowSize();
        const uint32_t rowSizeIn2 = in2.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t * in1Y = in1.data() + startY1   * rowSizeIn1 + startX1   * colorCount;
        const uint8_t * in2Y = in2.data() + startY2   * rowSizeIn2 + startX2   * colorCount;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut * colorCount;

        launchKernel2D( maximumCuda, width, height,
                        in1Y, rowSizeIn1, in2Y, rowSizeIn2, outY, rowSizeOut, width, height );
    }

    Image Minimum( const Image & in1, const Image & in2 )
    {
        Image_Function::ParameterValidation( in1, in2 );

        Image out( in1.width(), in1.height() );

        Minimum( in1, 0, 0, in2, 0, 0, out, 0, 0, out.width(), out.height() );

        return out;
    }

    void Minimum( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function::ParameterValidation( in1, in2, out );

        Minimum( in1, 0, 0, in2, 0, 0, out, 0, 0, out.width(), out.height() );
    }

    Image Minimum( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                   uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in1, startX1, startY1, in2, startX2, startY2, width, height );

        Image out( width, height );

        Minimum( in1, startX1, startY1, in2, startX2, startY2, out, 0, 0, out.width(), out.height() );

        return out;
    }

    void Minimum( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                  Image & out, uint32_t startXOut, uint32_t startYOut, uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in1, startX1, startY1, in2, startX2, startY2, out, startXOut, startYOut, width, height );

        const uint8_t colorCount = Image_Function::CommonColorCount( in1, in2, out );
        width = width * colorCount;

        const uint32_t rowSizeIn1 = in1.rowSize();
        const uint32_t rowSizeIn2 = in2.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t * in1Y = in1.data() + startY1   * rowSizeIn1 + startX1   * colorCount;
        const uint8_t * in2Y = in2.data() + startY2   * rowSizeIn2 + startX2   * colorCount;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut * colorCount;

        launchKernel2D( minimumCuda, width, height,
                        in1Y, rowSizeIn1, in2Y, rowSizeIn2, outY, rowSizeOut, width, height );
    }

    Image Subtract( const Image & in1, const Image & in2 )
    {
        Image_Function::ParameterValidation( in1, in2 );

        Image out( in1.width(), in1.height() );

        Subtract( in1, 0, 0, in2, 0, 0, out, 0, 0, out.width(), out.height() );

        return out;
    }

    void Subtract( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function::ParameterValidation( in1, in2, out );

        Subtract( in1, 0, 0, in2, 0, 0, out, 0, 0, out.width(), out.height() );
    }

    Image Subtract( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                    uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in1, startX1, startY1, in2, startX2, startY2, width, height );

        Image out( width, height );

        Subtract( in1, startX1, startY1, in2, startX2, startY2, out, 0, 0, out.width(), out.height() );

        return out;
    }

    void Subtract( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                   Image & out, uint32_t startXOut, uint32_t startYOut, uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in1, startX1, startY1, in2, startX2, startY2, out, startXOut, startYOut, width, height );

        const uint8_t colorCount = Image_Function::CommonColorCount( in1, in2, out );
        width = width * colorCount;

        const uint32_t rowSizeIn1 = in1.rowSize();
        const uint32_t rowSizeIn2 = in2.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t * in1Y = in1.data() + startY1   * rowSizeIn1 + startX1   * colorCount;
        const uint8_t * in2Y = in2.data() + startY2   * rowSizeIn2 + startX2   * colorCount;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut * colorCount;

        launchKernel2D( subtractCuda, width, height,
                        in1Y, rowSizeIn1, in2Y, rowSizeIn2, outY, rowSizeOut, width, height );
    }

    Image Threshold( const Image & in, uint8_t threshold )
    {
        Image_Function::ParameterValidation( in );

        Image out( in.width(), in.height() );

        Threshold( in, 0, 0, out, 0, 0, out.width(), out.height(), threshold );

        return out;
    }

    void Threshold( const Image & in, Image & out, uint8_t threshold )
    {
        Image_Function::ParameterValidation( in, out );

        Threshold( in, 0, 0, out, 0, 0, out.width(), out.height(), threshold );
    }

    Image Threshold( const Image & in, uint32_t startXIn, uint32_t startYIn, uint32_t width, uint32_t height, uint8_t threshold )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, width, height );

        Image out( width, height );

        Threshold( in, startXIn, startYIn, out, 0, 0, width, height, threshold );

        return out;
    }

    void Threshold( const Image & in, uint32_t startXIn, uint32_t startYIn, Image & out, uint32_t startXOut, uint32_t startYOut,
                    uint32_t width, uint32_t height, uint8_t threshold )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, out, startXOut, startYOut, width, height );
        Image_Function::VerifyGrayScaleImage( in, out );

        const uint32_t rowSizeIn  = in.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t * inY  = in.data()  + startYIn  * rowSizeIn  + startXIn;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut;

        launchKernel2D( thresholdCuda, width, height,
                        inY, rowSizeIn, outY, rowSizeOut, width, height, threshold );
    }

    Image Threshold( const Image & in, uint8_t minThreshold, uint8_t maxThreshold )
    {
        Image_Function::ParameterValidation( in );

        Image out( in.width(), in.height() );

        Threshold( in, 0, 0, out, 0, 0, out.width(), out.height(), minThreshold, maxThreshold );

        return out;
    }

    void Threshold( const Image & in, Image & out, uint8_t minThreshold, uint8_t maxThreshold )
    {
        Image_Function::ParameterValidation( in, out );

        Threshold( in, 0, 0, out, 0, 0, out.width(), out.height(), minThreshold, maxThreshold );
    }

    Image Threshold( const Image & in, uint32_t startXIn, uint32_t startYIn, uint32_t width, uint32_t height, uint8_t minThreshold,
                     uint8_t maxThreshold )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, width, height );

        Image out( width, height );

        Threshold( in, startXIn, startYIn, out, 0, 0, width, height, minThreshold, maxThreshold );

        return out;
    }

    void Threshold( const Image & in, uint32_t startXIn, uint32_t startYIn, Image & out, uint32_t startXOut, uint32_t startYOut,
                    uint32_t width, uint32_t height, uint8_t minThreshold, uint8_t maxThreshold )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, out, startXOut, startYOut, width, height );
        Image_Function::VerifyGrayScaleImage( in, out );

        const uint32_t rowSizeIn  = in.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t * inY  = in.data()  + startYIn  * rowSizeIn  + startXIn;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut;

        launchKernel2D( thresholdCuda, width, height,
                        inY, rowSizeIn, outY, rowSizeOut, width, height, minThreshold, maxThreshold );
    }
}
